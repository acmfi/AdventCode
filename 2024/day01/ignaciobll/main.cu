#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define SIZE 1000

#define gpuErrorCheck(ans, abort) { gpuAssert((ans), __FILE__, __LINE__, abort); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if(code != hipSuccess) {
    fprintf(stderr,"assert: %s %s %d\n", hipGetErrorString(code), file, line);
    if(abort) {
      exit(code);
    }
  }
}


int compare(const void *a, const void *b) {
  int *valA = (int*) a;
  int *valB = (int*) b;
  return *valA - *valB;
}


void parse_input(int* a, int* b) {
    FILE *file = fopen("input.txt", "r");

    char buffer[256];
    int line = 0;
    while (fgets(buffer, sizeof(buffer), file) != NULL && line < 1000) {
        sscanf(buffer, "%d %d", &a[line], &b[line]);
        line++;
    }

    fclose(file);
}

__global__ void diff(int *a, int *b, int *c) {
  // c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
  c[threadIdx.x] = abs(a[threadIdx.x] - b[threadIdx.x]);
}

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = SIZE * sizeof(int);

  gpuErrorCheck(hipMalloc((void **) &d_a, size), true);
  gpuErrorCheck(hipMalloc((void **) &d_b, size), true);
  gpuErrorCheck(hipMalloc((void **) &d_c, size), true);

  a = (int *) malloc(size);
  b = (int *) malloc(size);
  c = (int *) malloc(size);

  parse_input(a, b);

  qsort(a,SIZE,sizeof(int), compare);
  qsort(b,SIZE,sizeof(int), compare);

  // Copy input
  gpuErrorCheck(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice), true);
  gpuErrorCheck(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice), true);

  // Star1
  print_list(a, SIZE);
  print_list(b, SIZE);

  diff<<<1,SIZE>>>(d_a, d_b, d_c);
  gpuErrorCheck(hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost), true);

  print_list(c, SIZE);

  int acc = 0;
  for (int i = 0; i < SIZE; ++i) {
    acc += c[i];
  }

  printf("Star1: %d", acc);

  // Clean
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;

}
